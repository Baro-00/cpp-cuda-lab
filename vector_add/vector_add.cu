#include <iostream>
#include <hip/hip_runtime.h>

// Kernel CUDA - wykonywany na GPU
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    int N = 1024;
    size_t size = N * sizeof(float);

    // Alokacja pamięci na CPU
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Inicjalizacja danych wejściowych
    for (int i = 0; i < N; ++i) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Alokacja pamięci na GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Kopiowanie danych z CPU na GPU
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Ustawienia uruchamiania kernela
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Uruchomienie kernela CUDA
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Kopiowanie wyników z GPU na CPU
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Wypisanie kilku wyników
    std::cout << "Pierwsze 5 wynikow:\n";
    for (int i = 0; i < 5; ++i) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << "\n";
    }

    // Zwolnienie pamięci
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
